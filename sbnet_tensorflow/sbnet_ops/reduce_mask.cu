
#include <hip/hip_runtime.h>
/*

   Sparse Blocks Network
   Copyright (c) 2017, Uber Technologies, Inc.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

#ifdef GOOGLE_CUDA

#define EIGEN_USE_GPU
#define EIGEN_USE_THREADS

#include "reduce_mask.h"
#include "zero_block_counters.cu.h"
#include "reduce_mask.cu.h"
// #include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"
#include "cuda_helpers.h"

using namespace tensorflow;
using std::cout;
using std::endl;

// Define the GPU implementation that launches the CUDA kernel.
typedef Eigen::GpuDevice GPUDevice;
template <typename T> struct ReduceMaskFunctor<GPUDevice, T> {
    void operator()(const GPUDevice& d, // Device.
        const T* mask,                  // Mask array.
        int N,                          // Batch dimension of the mask.
        int H,                          // Height of the mask.
        int W,                          // Width of the mask.
        float threshold,                // Threshold for being active.
        int bOffsH0,                    // Block padding offset height, negative.
        int bOffsW0,                    // Block padding offset width, negative.
        int bSzH,                       // Block size height.
        int bSzW,                       // Block size width.
        int bStrH,                      // Block stride, height.
        int bStrW,                      // Block stride, width.
        int bCntH,                      // Number of blocks, height.
        int bCntW,                      // Number of blocks, width.
        unsigned int numBins,
        unsigned int binSize,
        long long* activeBlockIndices,  // Indices of active blocks.
        int* binCounts,                 // Number of indices of active blocks.
        bool avgPool
        )
    {
        zeroBlockCounters<<<24, 1024, 0, d.stream()>>>(numBins, (unsigned int*) binCounts);
        dim3 block(std::min(DIVUP(bSzH*bSzW, 32)*32, 1024), 1, 1);
        dim3 grid(bCntW, bCntH, N);
        reduceMask<<<grid, block, 0, d.stream()>>>(mask, N, H, W, // C is assumed to be 1
            threshold, // value to consider non-sparse block
            numBins,   // number of bins to partition activeBlockIndices to reduce atomics pressure
            binSize,
            (unsigned int*) binCounts, // counts for sub-blocks, initialized to 0
            (unsigned long long*) activeBlockIndices,
            bOffsH0,
            bOffsW0,      // generally negative - first block element offset for correct padding
            bSzH, bSzW,   // block sizes
            bStrH, bStrW, // block strides
            bCntH, bCntW, // block counts
            avgPool
            );
    }
};

// Instantiate functors for the types of OpKernels registered.
typedef Eigen::GpuDevice GPUDevice;
template struct ReduceMaskFunctor<GPUDevice, float>;

#endif // GOOGLE_CUDA
